#include "hip/hip_runtime.h"
#include "infini_train/include/device.h"
#include "infini_train/include/kernels/cuda/linear.h"

#include <memory>
#include <tuple>

#include "hipblas.h"
#include "glog/logging.h"

#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t status = call;                                                                                     \
        if (status != hipSuccess) {                                                                                   \
            LOG(FATAL) << "CUDA Error: " << hipGetErrorString(status) << " at " << __FILE__ << ":" << __LINE__;       \
        }                                                                                                              \
    } while (0)

#define CUBLAS_CHECK(call)                                                                                             \
    do {                                                                                                               \
        hipblasStatus_t status = call;                                                                                  \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                                                         \
            LOG(FATAL) << "CUBLAS Error: " << cublasGetStatusString(status) << " at " << __FILE__ << ":" << __LINE__;  \
        }                                                                                                              \
    } while (0)

std::shared_ptr<Tensor> LinearForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight,
                                      bool transpose, const std::shared_ptr<Tensor> &bias) {

    /*
        !transpose: output = input * weight + bias
        output[*, out_features] = input[*, in_features] * weight[in_features, out_features] + bias[out_features]

        transpose:  output = input * weight^T + bias
        output[*, out_features] = input[*, in_features] * weight[out_features, in_features]^T + bias[out_features]
    */

    CHECK_EQ(input->Dims().size(), 2);
    const int64_t bs = input->Dims()[0];
    const int64_t in_features = input->Dims()[1];
    CHECK_EQ(weight->Dims().size(), 2);

    // As for cublas:
    // C = alpha * op(B) * op(A) + beta * C
    // Dimensions:
    //   input:  (bs, in_features)
    //   weight: (in_features, out_features) or (out_features, in_features) if transposed
    //   output: (bs, out_features)
    int64_t out_features = 0;
    hipblasOperation_t op_weight = HIPBLAS_OP_N;

    if (transpose) {
        // weight: (out_features, in_features)
        CHECK_EQ(in_features, weight->Dims()[1]);
        out_features = weight->Dims()[0];
        op_weight = HIPBLAS_OP_T;
    } else {
        // weight: (in_features, out_features)
        CHECK_EQ(in_features, weight->Dims()[0]);
        out_features = weight->Dims()[1];
        op_weight = HIPBLAS_OP_N;
    }

    auto output = std::make_shared<Tensor>(std::vector<int64_t>{bs, out_features}, DataType::kFLOAT32,
                                           Device(DeviceType::kCUDA, 0));

    if (bias) {
        CHECK_EQ(bias->Dims().size(), 1);
        CHECK_EQ(bias->Dims()[0], out_features);
        for (int i = 0; i < bs; ++i) {
            hipMemcpy(static_cast<float *>(output->DataPtr()) + i * out_features, bias->DataPtr(),
                       out_features * sizeof(float), hipMemcpyDeviceToDevice);
        }
    } else {
        output->Fill<float>(0.0f);
    }

    const float alpha = 1.0f;
    const float beta = 1.0f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // C = alpha * op(B) * op(A) + beta * C
    // output = alpha * (input * weight) + beta * output
    hipblasSgemm(handle, op_weight, HIPBLAS_OP_N, out_features, bs, in_features, &alpha,
                static_cast<const float *>(weight->DataPtr()), (op_weight == HIPBLAS_OP_N) ? out_features : in_features,
                static_cast<const float *>(input->DataPtr()), in_features, &beta,
                static_cast<float *>(output->DataPtr()), out_features);

    hipblasDestroy(handle);

    return {output};
}

__global__ void set_ones(float *data, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        data[idx] = 1.0f;
    }
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
LinearBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight, bool transpose,
               int64_t out_features, const std::shared_ptr<Tensor> &grad_output, const bool bias) {
    CHECK_EQ(input->Dims().size(), 2);
    const int bs = input->Dims()[0];
    const int in_features = input->Dims()[1];
    CHECK_EQ(weight->Dims().size(), 2);

    auto grad_input = std::make_shared<Tensor>(input->Dims(), DataType::kFLOAT32, Device(DeviceType::kCUDA, 0));
    auto grad_weight = std::make_shared<Tensor>(weight->Dims(), DataType::kFLOAT32, Device(DeviceType::kCUDA, 0));
    grad_weight->Fill<float>(0.0f);
    std::shared_ptr<Tensor> grad_bias = nullptr;
    if (bias) {
        grad_bias = std::make_shared<Tensor>(std::vector<int64_t>{out_features}, DataType::kFLOAT32,
                                             Device(DeviceType::kCUDA, 0));
        grad_bias->Fill<float>(0.0f);
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    if (transpose) {
        // d_input = d_output * weight --> d_input.T = weight * d_output.T
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, in_features, bs, out_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), in_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_input->DataPtr()), in_features));

        // d_weight = d_output.T * input --> d_weight.T = input.T * d_output
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, out_features, in_features, bs, &alpha,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(grad_weight->DataPtr()), out_features));
    } else {
        // d_input = d_output * weight.T --> d_input.T = weight * d_output.T
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, in_features, bs, out_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), out_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_input->DataPtr()), in_features));

        // d_weight = input.T * d_output --> d_weight.T = d_output.T * input
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, out_features, in_features, bs, &alpha,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(grad_weight->DataPtr()), out_features));
    }

    // FIXME(dcj): remove this sync
    // CUDA_CHECK(hipDeviceSynchronize());

    // d_bias = \sum_i(i=0, bs-1) d_output[i]
    // TODO(dcj): use thrust::fill or reduce kernel do this
    if (bias) {
        auto ones
            = std::make_shared<Tensor>(std::vector<int64_t>{bs}, DataType::kFLOAT32, Device(DeviceType::kCUDA, 0));
        float *ones_ptr = static_cast<float *>(ones->DataPtr());

        int threads_per_block = 256;
        int num_blocks = (bs + threads_per_block - 1) / threads_per_block;

        set_ones<<<num_blocks, threads_per_block>>>(ones_ptr, bs);

        CUBLAS_CHECK(hipblasSgemv(
            handle, HIPBLAS_OP_N, out_features, bs, &alpha, static_cast<const float *>(grad_output->DataPtr()),
            out_features, static_cast<float *>(ones_ptr), 1, &beta, static_cast<float *>(grad_bias->DataPtr()), 1));
    }

    hipblasDestroy(handle);

    return {grad_input, grad_weight, grad_bias};
}
} // namespace infini_train::kernels::cuda
