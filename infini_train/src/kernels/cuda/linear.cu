#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "glog/logging.h"

#include "infini_train/include/dispatcher.h"
#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t status = call;                                                                                     \
        if (status != hipSuccess) {                                                                                   \
            LOG(FATAL) << "CUDA Error: " << hipGetErrorString(status) << " at " << __FILE__ << ":" << __LINE__;       \
        }                                                                                                              \
    } while (0)

#define CUBLAS_CHECK(call)                                                                                             \
    do {                                                                                                               \
        hipblasStatus_t status = call;                                                                                  \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                                                         \
            LOG(FATAL) << "CUBLAS Error: " << cublasGetStatusString(status) << " at " << __FILE__ << ":" << __LINE__;  \
        }                                                                                                              \
    } while (0)

std::shared_ptr<Tensor> MatmulForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &other) {
    /*
     output[*, m, n] = input[*, m, k] * other[*, k, n]
     */
    const auto &input_dims = input->Dims();
    const auto &other_dims = other->Dims();

    CHECK_GE(input_dims.size(), 2);
    CHECK_GE(other_dims.size(), 2);
    CHECK_EQ(input_dims.size(), other_dims.size());

    const int64_t m = input_dims[input_dims.size() - 2];
    const int64_t k = input_dims[input_dims.size() - 1];
    CHECK_EQ(k, other_dims[other_dims.size() - 2]);
    const int64_t n = other_dims[other_dims.size() - 1];

    const int64_t bs = std::accumulate(input_dims.rbegin() + 2, input_dims.rend(), 1, std::multiplies<int64_t>{});
    for (int64_t i = 0; i < input_dims.size() - 2; ++i) {
        CHECK_EQ(input_dims[i], other_dims[i]) << "Batch dims must match";
    }

    std::vector<int64_t> output_dims = input_dims;
    output_dims[output_dims.size() - 1] = n;
    auto output = std::make_shared<Tensor>(output_dims, DataType::kFLOAT32, input->GetDevice());

    const float alpha = 1.0f, beta = 0.0f;
    hipblasHandle_t handle;
    // TODO(zbl): create handle only once
    CUBLAS_CHECK(hipblasCreate(&handle));

    // cuBLAS is colmun-major
    // output = input * other --> output.T = other.T * input.T
    // C = A * B ==> output.T[*, n, m] = other.T[*, n, k] * input.T[*, k, m]
    // C = output.T[*, n, m]
    // A = other.T[*, n, k]
    // B = input.T[*, k, m]
    int lda = n;
    int ldb = k;
    int ldc = n;
    int64_t stride_a = n * k;
    int64_t stride_b = k * m;
    int64_t stride_c = m * n;
    // NOTE(zbl): the last hipblasGemmAlgo_t param has no effect on GPU arch >= sm_80(Ampere)
    CUBLAS_CHECK(hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, other->DataPtr(),
                                            HIP_R_32F, lda, stride_a, input->DataPtr(), HIP_R_32F, ldb, stride_b,
                                            &beta, output->DataPtr(), HIP_R_32F, ldc, stride_c, bs, HIP_R_32F,
                                            HIPBLAS_GEMM_DEFAULT));
    CUBLAS_CHECK(hipblasDestroy(handle));
    return output;
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
MatmulBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &other,
               const std::shared_ptr<Tensor> &grad_output) {
    /*
       grad_input[*, m, k] = grad_output[*, m, n] * other[*, k, n]^T
       grad_other[*, k, n] = input[*, m, k]^T * grad_output[*, m, n]
    */
    const auto &input_dims = input->Dims();
    const auto &other_dims = other->Dims();
    const auto &grad_output_dims = grad_output->Dims();

    CHECK_GE(input_dims.size(), 2);
    CHECK_EQ(input_dims.size(), other_dims.size());
    CHECK_EQ(input_dims.size(), grad_output_dims.size());

    const int64_t m = input_dims[input_dims.size() - 2];
    const int64_t k = input_dims[input_dims.size() - 1];
    const int64_t n = other_dims[other_dims.size() - 1];
    CHECK_EQ(k, other_dims[other_dims.size() - 2]);
    CHECK_EQ(m, grad_output_dims[grad_output_dims.size() - 2]);
    CHECK_EQ(n, grad_output_dims[grad_output_dims.size() - 1]);

    const int64_t bs = std::accumulate(input_dims.rbegin() + 2, input_dims.rend(), 1, std::multiplies<int64_t>{});
    for (int64_t i = 0; i < input_dims.size() - 2; ++i) {
        CHECK_EQ(input_dims[i], other_dims[i]) << "Batch dims must match";
        CHECK_EQ(input_dims[i], grad_output_dims[i]) << "Batch dims must match";
    }

    auto grad_input = std::make_shared<Tensor>(input_dims, DataType::kFLOAT32, grad_output->GetDevice());
    auto grad_other = std::make_shared<Tensor>(other_dims, DataType::kFLOAT32, grad_output->GetDevice());
    grad_input->Fill<float>(0.0f);
    grad_other->Fill<float>(0.0f);

    const float alpha = 1.0f, beta = 0.0f;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    {
        // cuBLAS is colmun-major
        // grad_input = grad_output * other.T --> grad_input.T = other * grad_output.T
        // C = A.T * B ==> grad_input.T[*, k, m] = other[*, k, n] * grad_output.T[*, n, m]
        // C = grad_input.T[*, k, m]
        // A = other.T[*, n, k]
        // B = grad_output.T[*, n, m]
        const int lda = n, ldb = n, ldc = k;
        const int64_t stride_a = k * n;
        const int64_t stride_b = n * m;
        const int64_t stride_c = m * k;
        CUBLAS_CHECK(hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, k, m, n, &alpha, other->DataPtr(),
                                                HIP_R_32F, lda, stride_a, grad_output->DataPtr(), HIP_R_32F, ldb,
                                                stride_b, &beta, grad_input->DataPtr(), HIP_R_32F, ldc, stride_c, bs,
                                                HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
    }

    {
        // cuBLAS is colmun-major
        // grad_other = input.T * grad_output --> grad_other.T =  grad_output.T * input
        // C = A * B.T ==> grad_other.T[*, n, k] = grad_output.T[*, n, m] * input[*, m, k]
        // C = grad_other.T[*, n, k]
        // A = grad_output.T[*, n, m]
        // B = input.T[*, k, m]
        const int lda = n, ldb = k, ldc = n;
        const int64_t stride_a = n * m;
        const int64_t stride_b = k * m;
        const int64_t stride_c = n * k;
        CUBLAS_CHECK(hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, k, m, &alpha,
                                                grad_output->DataPtr(), HIP_R_32F, lda, stride_a, input->DataPtr(),
                                                HIP_R_32F, ldb, stride_b, &beta, grad_other->DataPtr(), HIP_R_32F,
                                                ldc, stride_c, bs, HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
    }

    CUBLAS_CHECK(hipblasDestroy(handle));
    return {grad_input, grad_other};
}

__global__ void BiasCopyKernel(float *output, const float *bias, int bs, int out_features) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= bs * out_features) {
        return;
    }
    int j = idx % out_features;
    output[idx] = bias[j];
}

std::shared_ptr<Tensor> LinearForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight,
                                      bool transpose, const std::shared_ptr<Tensor> &bias) {

    /*
        !transpose: output = input * weight + bias
        output[*, out_features] = input[*, in_features] * weight[in_features, out_features] + bias[out_features]

        transpose:  output = input * weight^T + bias
        output[*, out_features] = input[*, in_features] * weight[out_features, in_features]^T + bias[out_features]
    */

    const auto &input_dims = input->Dims();
    CHECK_GE(input_dims.size(), 2);
    const int64_t bs = std::accumulate(input_dims.rbegin() + 1, input_dims.rend(), 1, std::multiplies<int64_t>{});
    const int64_t in_features = *input_dims.rbegin();

    const auto &weight_dims = weight->Dims();
    CHECK_EQ(weight_dims.size(), 2);
    CHECK_EQ(in_features, weight_dims[transpose ? 1 : 0]);

    // As for cublas:
    // C = alpha * op(B) * op(A) + beta * C
    // Dimensions:
    //   input:  (bs, in_features)
    //   weight: (in_features, out_features) or (out_features, in_features) if transposed
    //   output: (bs, out_features)
    const int64_t out_features = weight_dims[transpose ? 0 : 1];

    auto output_dims = input_dims;
    *output_dims.rbegin() = out_features;
    auto output = std::make_shared<Tensor>(output_dims, DataType::kFLOAT32, input->GetDevice());

    if (bias) {
        CHECK_EQ(bias->Dims().size(), 1);
        CHECK_EQ(bias->Dims()[0], out_features);
        int threads_per_block = 256;
        int num_blocks = (bs * out_features + threads_per_block - 1) / threads_per_block;
        BiasCopyKernel<<<num_blocks, threads_per_block>>>(
            static_cast<float *>(output->DataPtr()), static_cast<const float *>(bias->DataPtr()), bs, out_features);
    } else {
        output->Fill<float>(0.0f);
    }

    const float alpha = 1.0f;
    const float beta = 1.0f;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    // TODO(zbl): use hipblasSgemv if possible
    if (transpose) {
        // weight is [out_features, in_features] here

        // output = input * weight.T --> output.T = weight * input.T
        // C = output.T[out_features, bs]
        // A = weight.T[in_features, out_features]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, out_features, bs, in_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), in_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(output->DataPtr()), out_features));
    } else {
        // output = input * weight --> output.T =  weight.T * input.T
        // C = output.T[out_features, bs]
        // A = weight.T[out_features, in_features]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, out_features, bs, in_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), out_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(output->DataPtr()), out_features));
    }
    CUBLAS_CHECK(hipblasDestroy(handle));
    return output;
}

__global__ void set_ones(float *data, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        data[idx] = 1.0f;
    }
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
LinearBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight, bool transpose,
               int64_t out_features, const std::shared_ptr<Tensor> &grad_output, const bool bias) {
    const auto &input_dims = input->Dims();
    CHECK_GE(input_dims.size(), 2);
    const int64_t bs = std::accumulate(input_dims.rbegin() + 1, input_dims.rend(), 1, std::multiplies<int64_t>{});
    const int64_t in_features = *input_dims.rbegin();

    const auto &weight_dims = weight->Dims();
    CHECK_EQ(weight_dims.size(), 2);
    CHECK_EQ(in_features, weight_dims[transpose ? 1 : 0]);
    CHECK_EQ(out_features, weight_dims[transpose ? 0 : 1]);

    auto grad_input = std::make_shared<Tensor>(input_dims, DataType::kFLOAT32, grad_output->GetDevice());
    auto grad_weight = std::make_shared<Tensor>(weight_dims, DataType::kFLOAT32, grad_output->GetDevice());
    grad_input->Fill<float>(0.0f);
    grad_weight->Fill<float>(0.0f);
    std::shared_ptr<Tensor> grad_bias = nullptr;
    if (bias) {
        grad_bias = std::make_shared<Tensor>(std::vector<int64_t>{out_features}, DataType::kFLOAT32,
                                             grad_output->GetDevice());
        grad_bias->Fill<float>(0.0f);
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // TODO(zbl): use hipblasSgemv if possible
    if (transpose) {
        // weight is [out_features, in_features] here

        // d_input = d_output * weight --> d_input.T = weight.T * d_output.T
        // C = d_input.T[in_features, bs]
        // A = weight.T[in_features, out_features]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, in_features, bs, out_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), in_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_input->DataPtr()), in_features));

        // d_weight = d_output.T * input --> d_weight.T = input.T * d_output
        // C = d_weight.T[in_features, out_features]
        // A = input.T[in_features, bs]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, in_features, out_features, bs, &alpha,
                                 static_cast<const float *>(input->DataPtr()), in_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_weight->DataPtr()), in_features));
    } else {
        // weight is [in_features, out_features] here

        // d_input = d_output * weight.T --> d_input.T = weight * d_output.T
        // C = d_input.T[in_features, bs]
        // A = weight.T[out_features, in_features]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, in_features, bs, out_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), out_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_input->DataPtr()), in_features));

        // d_weight = input.T * d_output --> d_weight.T = d_output.T * input
        // C = d_weight.T[out_features, in_features]
        // A = d_output.T[out_features, bs]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, out_features, in_features, bs, &alpha,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(grad_weight->DataPtr()), out_features));
    }

    // d_bias = \sum_i(i=0, bs-1) d_output[i]
    // TODO(dcj): use thrust::fill or reduce kernel do this
    if (bias) {
        auto ones = std::make_shared<Tensor>(std::vector<int64_t>{bs}, DataType::kFLOAT32, grad_output->GetDevice());
        float *ones_ptr = static_cast<float *>(ones->DataPtr());

        int threads_per_block = 256;
        int num_blocks = (bs + threads_per_block - 1) / threads_per_block;

        set_ones<<<num_blocks, threads_per_block>>>(ones_ptr, bs);

        CUBLAS_CHECK(hipblasSgemv(
            handle, HIPBLAS_OP_N, out_features, bs, &alpha, static_cast<const float *>(grad_output->DataPtr()),
            out_features, static_cast<float *>(ones_ptr), 1, &beta, static_cast<float *>(grad_bias->DataPtr()), 1));
    }

    CUBLAS_CHECK(hipblasDestroy(handle));

    return {grad_input, grad_weight, grad_bias};
}
} // namespace infini_train::kernels::cuda

#define REGISTER_CUDA_LINEAR_KERNEL(kernel_name)                                                                       \
    REGISTER_KERNEL(infini_train::DeviceType::kCUDA, kernel_name, infini_train::kernels::cuda::kernel_name)

REGISTER_CUDA_LINEAR_KERNEL(MatmulForward)
REGISTER_CUDA_LINEAR_KERNEL(MatmulBackward)
REGISTER_CUDA_LINEAR_KERNEL(LinearForward)
REGISTER_CUDA_LINEAR_KERNEL(LinearBackward)

#undef REGISTER_CUDA_LINEAR_KERNEL
