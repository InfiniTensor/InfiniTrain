#include "hip/hip_runtime.h"
#include "infini_train/include/device.h"
#include "infini_train/include/kernels/cuda/linear.h"

#include <memory>
#include <tuple>

#include "hipblas.h"
#include "glog/logging.h"

#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t status = call;                                                                                     \
        if (status != hipSuccess) {                                                                                   \
            LOG(FATAL) << "CUDA Error: " << hipGetErrorString(status) << " at " << __FILE__ << ":" << __LINE__;       \
        }                                                                                                              \
    } while (0)

#define CUBLAS_CHECK(call)                                                                                             \
    do {                                                                                                               \
        hipblasStatus_t status = call;                                                                                  \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                                                         \
            LOG(FATAL) << "CUBLAS Error: " << cublasGetStatusString(status) << " at " << __FILE__ << ":" << __LINE__;  \
        }                                                                                                              \
    } while (0)

std::shared_ptr<Tensor> LinearForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight,
                                      const std::shared_ptr<Tensor> &bias) {
    CHECK_EQ(input->Dims().size(), 2);
    const int bs = input->Dims()[0];
    const int in_feature = input->Dims()[1];
    CHECK_EQ(weight->Dims().size(), 2);
    CHECK_EQ(in_feature, weight->Dims()[0]);
    const int out_feature = weight->Dims()[1];

    auto output = std::make_shared<Tensor>(std::vector<int64_t>{bs, out_feature}, DataType::kFLOAT32,
                                           Device(DeviceType::kCUDA, 0));

    if (bias) {
        CHECK_EQ(bias->Dims().size(), 1);
        CHECK_EQ(bias->Dims()[0], out_feature);
        for (int i = 0; i < bs; ++i) {
            hipMemcpy(static_cast<float *>(output->DataPtr()) + i * out_feature, bias->DataPtr(),
                       out_feature * sizeof(float), hipMemcpyDeviceToDevice);
        }
    } else {
        output->Fill<float>(0.0f);
    }

    const float alpha = 1.0f;
    const float beta = 1.0f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // output = alpha * (input * weight) + beta * output
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, out_feature, bs, in_feature, &alpha,
                static_cast<const float *>(weight->DataPtr()), out_feature,
                static_cast<const float *>(input->DataPtr()), in_feature, &beta,
                static_cast<float *>(output->DataPtr()), out_feature);

    hipblasDestroy(handle);

    return {output};
}

__global__ void set_ones(float *data, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        data[idx] = 1.0f;
    }
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
LinearBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight,
               const std::shared_ptr<Tensor> &bias, const std::shared_ptr<Tensor> &grad_output) {
    CHECK_EQ(input->Dims().size(), 2);
    const int bs = input->Dims()[0];
    const int in_feature = input->Dims()[1];
    CHECK_EQ(weight->Dims().size(), 2);
    CHECK_EQ(in_feature, weight->Dims()[0]);
    const int out_feature = weight->Dims()[1];
    if (bias) {
        CHECK_EQ(bias->Dims().size(), 1);
        CHECK_EQ(bias->Dims()[0], out_feature);
    }
    auto grad_input = std::make_shared<Tensor>(input->Dims(), DataType::kFLOAT32, Device(DeviceType::kCUDA, 0));
    auto grad_weight = std::make_shared<Tensor>(weight->Dims(), DataType::kFLOAT32, Device(DeviceType::kCUDA, 0));
    grad_weight->Fill<float>(0.0f);
    std::shared_ptr<Tensor> grad_bias = nullptr;
    if (bias) {
        grad_bias = std::make_shared<Tensor>(bias->Dims(), DataType::kFLOAT32, Device(DeviceType::kCUDA, 0));
        grad_bias->Fill<float>(0.0f);
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // d_input = d_output * weight^T --> d_input^T = weight * d_output^T
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, in_feature, bs, out_feature, &alpha,
                             static_cast<const float *>(weight->DataPtr()), out_feature,
                             static_cast<const float *>(grad_output->DataPtr()), out_feature, &beta,
                             static_cast<float *>(grad_input->DataPtr()), in_feature));

    // d_weight = input^T * d_output --> d_weight^T = d_output^T * input
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, out_feature, in_feature, bs, &alpha,
                             static_cast<const float *>(grad_output->DataPtr()), out_feature,
                             static_cast<const float *>(input->DataPtr()), in_feature, &beta,
                             static_cast<float *>(grad_weight->DataPtr()), out_feature));
    // FIXME(dcj): remove this sync
    CUDA_CHECK(hipDeviceSynchronize());

    // d_bias = \sum_i(i=0, bs-1) d_output[i]
    // TODO(dcj): use thrust::fill or reduce kernel do this
    if (bias) {
        auto ones
            = std::make_shared<Tensor>(std::vector<int64_t>{bs}, DataType::kFLOAT32, Device(DeviceType::kCUDA, 0));
        float *ones_ptr = static_cast<float *>(ones->DataPtr());

        int threads_per_block = 256;
        int num_blocks = (bs + threads_per_block - 1) / threads_per_block;

        set_ones<<<num_blocks, threads_per_block>>>(ones_ptr, bs);

        CUBLAS_CHECK(hipblasSgemv(
            handle, HIPBLAS_OP_N, out_feature, bs, &alpha, static_cast<const float *>(grad_output->DataPtr()),
            out_feature, static_cast<float *>(ones_ptr), 1, &beta, static_cast<float *>(grad_bias->DataPtr()), 1));
    }

    hipblasDestroy(handle);

    return {grad_input, grad_weight, grad_bias};
}
} // namespace infini_train::kernels::cuda
