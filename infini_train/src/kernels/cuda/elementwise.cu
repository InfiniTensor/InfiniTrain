#include "hip/hip_runtime.h"
#include "infini_train/include/kernels/cuda/elementwise.h"

#include <cmath>
#include <cstdint>
#include <functional>
#include <memory>
#include <utility>

#include "glog/logging.h"

#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

namespace {

template <typename T, typename Func>
__global__ void UnaryForwardKernel(T *output, Func fn, size_t num_elements, size_t offset, const T *input) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (idx < num_elements) {
        output[idx] = fn(input[idx]);
    }
}

template <typename T, typename Func>
__global__ void BinaryForwardKernel(T *output, Func fn, size_t num_elements_a, size_t num_elements_b, size_t offset,
                                    const T *a, const T *b) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (idx < num_elements_a) {
        output[idx] = fn(a[idx], b[idx % num_elements_b]);
    }
}

// launch the given kernel function with the given output and inputs
template <size_t BLOCK_SIZE, typename T, typename Kernel, typename... Inputs>
void LaunchKernel(Kernel &&kernel, const std::shared_ptr<Tensor> &output, const Inputs &...inputs) {
    auto extract_ptrs
        = [](const auto &...ts) { return std::make_tuple(static_cast<T *>(ts ? ts->DataPtr() : nullptr)...); };
    auto input_ptrs = extract_ptrs(inputs...);

    const size_t num_elements = output->NumElements();
    dim3 block_dims(std::min(BLOCK_SIZE, static_cast<size_t>(1024)));
    dim3 grid_dims(CEIL_DIV(num_elements, block_dims.x));
    const size_t step = grid_dims.x * block_dims.x;

    for (size_t offset = 0; offset < num_elements; offset += step) {
        std::apply([&](auto... ptrs) { kernel(grid_dims, block_dims, offset, ptrs...); }, input_ptrs);
    }
}

// launch a forward elementwise operation given the calculation function, output, and the inputs
// Note: currently only support unary and binary operations
template <size_t BLOCK_SIZE, typename T, typename Func, typename... Inputs>
void LaunchForward(Func func, const std::shared_ptr<Tensor> &output, const Inputs &...inputs) {
    T *output_ptr = static_cast<T *>(output->DataPtr());

    if constexpr (sizeof...(inputs) == 1) {
        // Unary case
        LaunchKernel<BLOCK_SIZE, T>(
            [&](dim3 grid, dim3 block, size_t offset, auto... ptrs) {
                UnaryForwardKernel<<<grid, block>>>(output_ptr, func, output->NumElements(), offset, ptrs...);
            },
            output, inputs...);
    } else if constexpr (sizeof...(inputs) == 2) {
        // Binary case
        auto input_tuple = std::make_tuple(inputs...);
        const auto &input_a = std::get<0>(input_tuple);
        const auto &input_b = std::get<1>(input_tuple);

        LaunchKernel<BLOCK_SIZE, T>(
            [&](dim3 grid, dim3 block, size_t offset, const T *a_ptr, const T *b_ptr) {
                BinaryForwardKernel<<<grid, block>>>(output_ptr, func, input_a->NumElements(), input_b->NumElements(),
                                                     offset, a_ptr, b_ptr);
            },
            output, inputs...);
    } else {
        static_assert(sizeof...(inputs) == 1 || sizeof...(inputs) == 2,
                      "LaunchForward currently only supports unary and binary operations.");
    }
}

// Backward kernel for unary operators
template <typename T, typename Func>
__global__ void UnaryBackwardKernel(T *output, Func fn, size_t num_elements, size_t offset, const T *grad_output,
                                    const T *input) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (idx < num_elements) {
        output[idx] = grad_output[idx] * fn(input ? input[idx] : T(0));
    }
}

// Backward kernel for binary operators
template <typename T, typename FuncA, typename FuncB>
__global__ void BinaryBackwardKernel(T *output_a, T *output_b, FuncA fun_a, FuncB fun_b, int64_t a_num_elements,
                                     int64_t b_num_elements, size_t offset, const T *grad_output, const T *input_a,
                                     const T *input_b) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (idx < a_num_elements) {
        const T &a = input_a ? input_a[idx] : T(0);
        const T &b = input_b ? input_b[idx % b_num_elements] : T(0);
        output_a[idx] = grad_output[idx] * fun_a(a, b);
        atomicAdd(&output_b[idx % b_num_elements], grad_output[idx] * fun_b(a, b));
    }
}

// launch unary operator's backward kernel
template <size_t BLOCK_SIZE, typename T, typename Func, typename... Inputs>
void LaunchBackward(Func func, const std::shared_ptr<Tensor> &output, const std::shared_ptr<Tensor> &grad_output,
                    const Inputs &...inputs) {
    T *output_ptr = static_cast<T *>(output->DataPtr());
    const T *grad_ptr = static_cast<const T *>(grad_output->DataPtr());

    LaunchKernel<BLOCK_SIZE, T>(
        [=](dim3 grid, dim3 block, size_t offset, auto... ptrs) {
            UnaryBackwardKernel<<<grid, block>>>(output_ptr, func, output->NumElements(), offset, grad_ptr, ptrs...);
        },
        output, inputs...);
}

// launch binary operator's backward kernel
template <size_t BLOCK_SIZE, typename T, typename FuncA, typename FuncB, typename... Inputs>
void LaunchBackward(FuncA fun_a, FuncB fun_b, const std::shared_ptr<Tensor> &output_a,
                    const std::shared_ptr<Tensor> &output_b, int64_t a_num_elements, int64_t b_num_elements,
                    const std::shared_ptr<Tensor> &grad_output, const Inputs &...inputs) {
    T *output_a_ptr = static_cast<T *>(output_a->DataPtr());
    T *output_b_ptr = static_cast<T *>(output_b->DataPtr());
    const T *grad_output_ptr = static_cast<const T *>(grad_output->DataPtr());
    LaunchKernel<BLOCK_SIZE, T>(
        [=](dim3 grid, dim3 block, size_t offset, auto... ptrs) {
            BinaryBackwardKernel<<<grid, block>>>(output_a_ptr, output_b_ptr, fun_a, fun_b, a_num_elements,
                                                  b_num_elements, offset, grad_output_ptr, ptrs...);
        },
        output_a, inputs...);
}

template <typename Func> std::shared_ptr<Tensor> UnaryForward(const std::shared_ptr<Tensor> &input, Func unary_fn) {
    auto dtype = input->Dtype();
    auto output = std::make_shared<Tensor>(input->Dims(), dtype, input->GetDevice());

    switch (dtype) {
    case DataType::kFLOAT32:
        LaunchForward<256, float>(unary_fn, output, input);
        break;
    default:
        LOG(FATAL) << "CUDA unary forward: 'Unsupported data type' at " << __FILE__ << ":" << __LINE__;
    }

    return output;
}

template <typename Func>
std::shared_ptr<Tensor> UnaryBackward(const std::shared_ptr<Tensor> &grad_output, const std::shared_ptr<Tensor> &a,
                                      Func unary_fn) {
    auto dtype = grad_output->Dtype();
    auto output = std::make_shared<Tensor>(grad_output->Dims(), dtype, grad_output->GetDevice());
    output->Fill<float>(0.0f);
    switch (dtype) {
    case DataType::kFLOAT32:
        LaunchBackward<256, float>(unary_fn, output, grad_output, a);
        break;
    default:
        LOG(FATAL) << "CUDA unary backward: 'Unsupported data type' at " << __FILE__ << ":" << __LINE__;
    }

    return output;
}

template <typename Func>
std::shared_ptr<Tensor> BinaryForward(const std::shared_ptr<Tensor> &a, const std::shared_ptr<Tensor> &b,
                                      Func binary_fn) {
    auto dtype = a->Dtype();
    // Currently a and b should have the same data type and only one-way broadcasting from b to a is assumed by default
    CHECK(dtype == b->Dtype() && a->NumElements() >= b->NumElements() && a->NumElements() % b->NumElements() == 0);

    auto output = std::make_shared<Tensor>(a->Dims(), dtype, a->GetDevice());

    switch (dtype) {
    case DataType::kFLOAT32:
        LaunchForward<256, float>(binary_fn, output, a, b);
        break;
    default:
        LOG(FATAL) << "CUDA binary forward: 'Unsupported data type' at " << __FILE__ << ":" << __LINE__;
    }

    return output;
}

template <typename FuncA, typename FuncB>
std::pair<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
BinaryBackward(const std::shared_ptr<Tensor> &grad_output, const std::shared_ptr<Tensor> &a,
               const std::shared_ptr<Tensor> &b, const std::vector<int64_t> &a_dims, const std::vector<int64_t> &b_dims,
               FuncA fn_a, FuncB fn_b) {
    const auto a_num_elements = std::accumulate(a_dims.begin(), a_dims.end(), 1, std::multiplies<int64_t>());
    const auto b_num_elements = std::accumulate(b_dims.begin(), b_dims.end(), 1, std::multiplies<int64_t>());

    CHECK(a_num_elements >= b_num_elements && a_num_elements % b_num_elements == 0);
    if (a) {
        CHECK(a_num_elements == a->NumElements());
    }
    if (b) {
        CHECK(b_num_elements == b->NumElements());
    }
    auto dtype = grad_output->Dtype();
    auto device = grad_output->GetDevice();

    // Currently a and b should have the same data type
    if (a && b) {
        CHECK(a->Dtype() == b->Dtype());
    }
    auto grad_a = std::make_shared<Tensor>(a_dims, dtype, device);
    auto grad_b = std::make_shared<Tensor>(b_dims, dtype, device);
    grad_a->Fill<float>(0.0f);
    grad_b->Fill<float>(0.0f);
    switch (dtype) {
    case DataType::kFLOAT32:
        LaunchBackward<256, float>(fn_a, fn_b, grad_a, grad_b, a_num_elements, b_num_elements, grad_output, a, b);
        break;
    default:
        LOG(FATAL) << "CUDA binary backward: 'Unsupported data type' at " << __FILE__ << ":" << __LINE__;
    }

    return {grad_a, grad_b};
}
} // namespace

std::shared_ptr<Tensor> TanhForward(const std::shared_ptr<Tensor> &input) {
    return UnaryForward(input, [] __device__(float x) { return tanhf(x); });
}

std::shared_ptr<Tensor> TanhBackward(const std::shared_ptr<Tensor> &grad_output,
                                     const std::shared_ptr<Tensor> &output) {
    return UnaryBackward(grad_output, output, [] __device__(float x) { return 1.0 - x * x; });
}

std::shared_ptr<Tensor> PowForward(const std::shared_ptr<Tensor> &input, float exponent) {
    return UnaryForward(input, [exponent] __device__(float x) { return powf(x, exponent); });
}

std::shared_ptr<Tensor> PowBackward(const std::shared_ptr<Tensor> &grad_output, const std::shared_ptr<Tensor> &input,
                                    float exponent) {
    return UnaryBackward(grad_output, input,
                         [exponent] __device__(float x) { return exponent * powf(x, exponent - 1.0f); });
}

std::shared_ptr<Tensor> EqualsScalarForward(const std::shared_ptr<Tensor> &a, float scalar) {
    return UnaryForward(a, [scalar] __device__(float x) { return x == scalar ? 1.0f : 0.0f; });
}

std::shared_ptr<Tensor> AddForward(const std::shared_ptr<Tensor> &a, const std::shared_ptr<Tensor> &b) {
    return BinaryForward(a, b, [] __device__(float x, float y) { return x + y; });
}

__global__ void AddBackwardReduceKernel(const float *grad_output, float *grad_b, const int64_t *out_strides,
                                        const int64_t *out_dims, int ndim, const int64_t *b_strides,
                                        const int64_t *b_dims, int b_ndim, int64_t num_elements) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_elements) {
        return;
    }

    int64_t tmp = idx;
    int64_t out_index[16]; // Assume ndim < 16
    for (int i = 0; i < ndim; ++i) {
        out_index[i] = tmp / out_strides[i];
        tmp %= out_strides[i];
    }

    int64_t b_offset = 0;
    for (int i = 0; i < b_ndim; ++i) {
        int out_axis = ndim - b_ndim + i;
        int64_t idx_in_b;
        if (out_axis < 0 || b_dims[i] == 1) {
            idx_in_b = 0;
        } else {
            idx_in_b = out_index[out_axis];
        }
        b_offset += idx_in_b * b_strides[i];
    }

    atomicAdd(&grad_b[b_offset], grad_output[idx]);
}

std::pair<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>> AddBackward(const std::shared_ptr<Tensor> &grad_output,
                                                                        const std::vector<int64_t> &a_dims,
                                                                        const std::vector<int64_t> &b_dims) {
    // TODO(zbl): assume ndim <= 16 here
    CHECK_LE(grad_output->Dims().size(), 16);
    CHECK_EQ(a_dims.size(), grad_output->Dims().size());

    auto grad_a = std::make_shared<Tensor>(a_dims, DataType::kFLOAT32, grad_output->GetDevice());
    hipMemcpyAsync(grad_a->DataPtr(), grad_output->DataPtr(), grad_output->NumElements() * sizeof(float),
                    hipMemcpyDeviceToDevice, 0);

    auto grad_b = std::make_shared<Tensor>(b_dims, DataType::kFLOAT32, grad_output->GetDevice());
    grad_b->Fill<float>(0.0f);

    const auto &out_dims = grad_output->Dims();
    const int ndim = out_dims.size();
    const int b_ndim = b_dims.size();
    const int64_t num_elements = grad_output->NumElements();

    std::vector<int64_t> out_strides(ndim);
    if (ndim > 0) {
        out_strides[ndim - 1] = 1;
        for (int i = ndim - 2; i >= 0; --i) { out_strides[i] = out_strides[i + 1] * out_dims[i + 1]; }
    }

    std::vector<int64_t> b_strides(b_ndim);
    if (b_ndim > 0) {
        b_strides[b_ndim - 1] = 1;
        for (int i = b_ndim - 2; i >= 0; --i) { b_strides[i] = b_strides[i + 1] * b_dims[i + 1]; }
    }

    int64_t *d_out_strides = nullptr;
    int64_t *d_out_dims = nullptr;
    int64_t *d_b_strides = nullptr;
    int64_t *d_b_dims = nullptr;

    hipMallocAsync(&d_out_strides, 2 * (ndim + b_ndim) * sizeof(*d_out_strides), 0);
    d_out_dims = d_out_strides + ndim;
    d_b_strides = d_out_dims + ndim;
    d_b_dims = d_b_strides + b_ndim;

    hipMemcpyAsync(d_out_strides, out_strides.data(), ndim * sizeof(int64_t), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(d_out_dims, out_dims.data(), ndim * sizeof(int64_t), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(d_b_strides, b_strides.data(), b_ndim * sizeof(int64_t), hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(d_b_dims, b_dims.data(), b_ndim * sizeof(int64_t), hipMemcpyHostToDevice, 0);

    int threads = 256;
    int blocks = (num_elements + threads - 1) / threads;

    AddBackwardReduceKernel<<<blocks, threads>>>(static_cast<const float *>(grad_output->DataPtr()),
                                                 static_cast<float *>(grad_b->DataPtr()), d_out_strides, d_out_dims,
                                                 ndim, d_b_strides, d_b_dims, b_ndim, num_elements);
    // NOTE(zbl): hipFree() needs explicit sync when hipMallocAsync() is called
    hipFreeAsync(d_out_strides, 0);

    return {grad_a, grad_b};
}

std::shared_ptr<Tensor> AddScalarForward(const std::shared_ptr<Tensor> &a, float scalar) {
    return UnaryForward(a, [scalar] __device__(float x) { return x + scalar; });
}

std::shared_ptr<Tensor> AddScalarBackward(const std::shared_ptr<Tensor> &grad_output) {
    return UnaryBackward(grad_output, nullptr, [] __device__(float) { return 1.0f; });
}

std::shared_ptr<Tensor> MulForward(const std::shared_ptr<Tensor> &a, const std::shared_ptr<Tensor> &b) {
    return BinaryForward(a, b, [] __device__(float x, float y) { return x * y; });
}

std::pair<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>> MulBackward(const std::shared_ptr<Tensor> &a,
                                                                        const std::shared_ptr<Tensor> &b,
                                                                        const std::shared_ptr<Tensor> &grad_output) {
    return BinaryBackward(
        grad_output, a, b, a->Dims(), b->Dims(), [] __device__(float, float y) { return y; },
        [] __device__(float x, float) { return x; });
}

std::shared_ptr<Tensor> MulScalarForward(const std::shared_ptr<Tensor> &a, float scalar) {
    return UnaryForward(a, [scalar] __device__(float x) { return x * scalar; });
}

std::shared_ptr<Tensor> MulScalarBackward(const std::shared_ptr<Tensor> &grad_output, float scalar) {
    return UnaryBackward(grad_output, nullptr, [scalar] __device__(float) { return scalar; });
}
} // namespace infini_train::kernels::cuda
