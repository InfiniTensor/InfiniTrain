#include "hip/hip_runtime.h"
#include "infini_train/include/kernels/cuda/accumulate_grad.h"

#include <memory>

#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

__global__ void AccumulateGradKernel(const float *grad_ptr, float rate, float *tensor_ptr, size_t num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        tensor_ptr[idx] += rate * grad_ptr[idx];
    }
}

void AccumulateGrad(const std::shared_ptr<Tensor> &gradient, float rate, const std::shared_ptr<Tensor> &tensor) {
    size_t num_elements = gradient->NumElements();

    const float *grad_ptr = reinterpret_cast<const float *>(gradient->DataPtr());
    float *tensor_ptr = reinterpret_cast<float *>(tensor->DataPtr());

    int threads_per_block = 256;
    int num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;

    AccumulateGradKernel<<<num_blocks, threads_per_block>>>(grad_ptr, rate, tensor_ptr, num_elements);
}
} // namespace infini_train::kernels::cuda
